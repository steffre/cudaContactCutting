#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include "../Header/functions.h"

__global__ void AddInts(double *p, int nodes, double sp, int nx, int ny, int nz){
    int ind1x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int ind2y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int ind3z = (blockIdx.z * blockDim.z) + threadIdx.z;
    int thn= ind1x+nx*ind2y+nx*ny*ind3z;

    if (ind1x < nx && ind2y<ny && ind3z<nz){
        p[thn*3 + 0] = ind1x*sp;
        p[thn*3 + 1] = ind2y*sp;
        p[thn*3 + 2] = ind3z*sp;
    }
}


int nodeGrid(int nodes, int nx, int ny, int nz, double *p, double sp){

    double *d_p;   /* p matrix that has the coordnate of the nodes as pointer*/

    //std::cout<<"\n nodes size : "<<nodes<<"\n";

    hipError_t cudaStatus;
    cudaStatus = hipMalloc(&d_p,sizeof(double) * nodes * 3);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_p! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_p,p, sizeof(double) * nodes * 3, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_p into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_p);
        return 1;
    }

    /*louncing the kernel*/
    int threadsX = 16;                                   /*Thread per blocks on x dir */
    int threadsY = 16;                                   /*Thread per blocks on y dir */ 
    int threadsZ = 4;                                    /*Thread per blocks on z dir */ 
    int blocksX = nx/threadsX +1;                        /*Blocks on the x dir*/
    int blocksY = ny/threadsY +1;                        /*Blocks on the y dir*/
    int blocksZ = nz/threadsY +1;                        /*Blocks on the z dir*/

    dim3 THREADS( threadsX ,threadsY,threadsZ);
    dim3 BLOCKS(blocksX,blocksY,blocksZ);

    AddInts<<<THREADS, BLOCKS>>>(d_p,nodes,sp,nx,ny,nz);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching AddInts!\n";
        return 1;
    }

    cudaStatus = hipMemcpy(p,d_p, sizeof(double) * nodes * 3, hipMemcpyDeviceToHost) ;
    if(cudaStatus!= hipSuccess){
        delete[] p;
        hipFree(d_p);
        std:: cout<<" \n Could not copy d_p back into the cpu !";
        std::cerr << "hipMemcpy device to host failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "AddInts launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    hipFree(d_p);

    return 0;
}
