#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include "../Header/functions.h"
#include <cmath>

__global__ void inE(int nodes, int *Nncon, int maxncon, double *d_energy){
    int node = (blockIdx.x * blockDim.x) + threadIdx.x;
    int idxncon = node*maxncon;
    
    if (node < nodes){
        
        for (int j=0;j<Nncon[node];j++){
            d_energy[idxncon + j ] = 0;
            
        }
    }
}


int iniEnergy(int nodes, int *Nncon, int maxncon, double *energy){

    double *d_energy;    /* p matrix that has the coordnate of the nodes as pointer */
    int *d_Nncon;   /* vector that tells for ech node how many nodes are connected */

    hipError_t cudaStatus;
    cudaStatus = hipMalloc(&d_energy,sizeof(double) * nodes * maxncon);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_energy! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_Nncon,sizeof(int) * nodes);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_Nncon! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_Nncon,Nncon, sizeof(int) * nodes, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy Nncon into d_Nncon into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_Nncon);
        return 1;
    }

    cudaStatus = hipMemset(d_energy, 0, sizeof(double) * nodes * maxncon);
    if(cudaStatus != hipSuccess){
        std::cerr << "hipMemset failed for d_energy: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    /*louncing the kernel*/
    int threads = 512;                                   /*Thread per blocks on x dir */
    int blocks = (nodes+threads-1)/threads;                        /*Blocks on the x dir*/

    inE<<< blocks, threads >>>(nodes,d_Nncon,maxncon,d_energy);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "inD kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        // Handle error, free memory, etc.
        return 1;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching inD!\n";
        // Handle error, free memory, etc.
        return 1;
    }
    
    //gpuErrchk(hipPeekAtLastError()); 
    //gpuErrchk(hipDeviceSynchronize());

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching nodCon!\n";
        return 1;
    }

    cudaStatus = hipMemcpy(energy, d_energy, sizeof(int) * nodes*maxncon, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        delete[] energy;
        hipFree(d_energy);
        hipFree(d_Nncon);
        std::cerr << "Failed to copy d_damage from device to host: " << hipGetErrorString(cudaStatus) << "\n";
        return 1;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "nodCon launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }


    
    hipFree(d_Nncon);
    hipFree(d_energy);

    return 0;
}
