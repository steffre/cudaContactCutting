#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include "../Header/functions.h"

__global__ void updateNodeskernel(double *Fnodes,double *p, double *pnew, int nodes, double deltat){
    
    int nodeidx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (nodeidx < nodes * 3){

        pnew[nodeidx] = p[nodeidx]+Fnodes[nodeidx]*deltat;
        
    }
}


int updateNodes(double *Fnodes,double *p,double *pnew, double deltat, int nodes){

    double *d_Fnodes;       /* Vector of the forces */
    double *d_p;            /* Vector of the position of the nodes */
    double *d_pnew;         /* New posiiton after apply force */
    

    hipError_t cudaStatus;
    cudaStatus = hipMalloc(&d_Fnodes,sizeof(double) * nodes * 3);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_Fnodes! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_p,sizeof(double) * nodes * 3);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_p! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_pnew,sizeof(double) * nodes * 3);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_pnew! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_Fnodes,Fnodes, sizeof(double) * nodes * 3, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy Fnodes into d_Fnodes into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_Fnodes);
        return 1;
    }

    cudaStatus = hipMemcpy(d_p,p, sizeof(double) * nodes * 3, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_TopNodes into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_p);
        return 1;
    }

   

    /*louncing the kernel*/
    int threads = 1024;                                   /* Thread per blocks on x dir */
    int blocks = (nodes*3)/threads+1;              /* Blocks on the x dir*/

    //std::cout << "Nodes: " << nodes << std::endl;
    //std::cout << "Threads: " << threads << std::endl;
    //std::cout << "Blocks: " << blocks << std::endl;


    updateNodeskernel<<< blocks, threads >>>(d_Fnodes,d_p,d_pnew,nodes,deltat);

    //gpuErrchk(hipPeekAtLastError()); 
    //gpuErrchk(hipDeviceSynchronize());

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching AddInts!\n";
        return 1;
    }

    cudaStatus = hipMemcpy(pnew,d_pnew, sizeof(double) * nodes * 3, hipMemcpyDeviceToHost) ;
    if(cudaStatus!= hipSuccess){
        delete[] pnew;
        hipFree(d_pnew);
        std:: cout<<" \n Could not copy d_p back into the cpu !";
        std::cerr << "hipMemcpy device to host failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "AddInts launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }


    hipFree(d_Fnodes);
    hipFree(d_p);
    hipFree(d_pnew);

    return 0;
}
