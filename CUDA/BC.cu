#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include "../Header/functions.h"

__global__ void BCdisp(double *p,int *TopNodes, int *BottomNodes, int nodes, double disp, double yh){
    
    int nodeidx = (blockIdx.x * blockDim.x) + threadIdx.x;

    int pIdx_y = nodeidx * 3 + 1;
    

    if (nodeidx < nodes){

        /*this is finding all the nodes in p that has the y position higher than yh - BoundSize */
        if (TopNodes[nodeidx] == 1){
            p[pIdx_y] = p[pIdx_y]+p[pIdx_y]*disp/yh;
        }
        if(BottomNodes[nodeidx] == 1){
            p[pIdx_y] = p[pIdx_y]+p[pIdx_y]*disp/yh;
        }
        
    }
}


int BC(int nodes, double *p,int *TopNodes, int *BottomNodes, double disp, double yh){

    double *d_p;            /* p matrix that has the coordnate of the nodes as pointer*/
    int *d_TopNodes;        /* TopNodes pointer to the memory that has top nodes saved*/
    int *d_BottomNodes;     /* TopNodes pointer to the memory that has top nodes saved*/
    

    hipError_t cudaStatus;
    cudaStatus = hipMalloc(&d_p,sizeof(double) * nodes * 3);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_p! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_TopNodes,sizeof(int) * nodes);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_TopNodes! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_BottomNodes,sizeof(int) * nodes);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_BottomNodes! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_p,p, sizeof(double) * nodes * 3, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_p into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_p);
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_TopNodes,TopNodes, sizeof(int) * nodes, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_TopNodes into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_p);
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_BottomNodes,BottomNodes, sizeof(int) * nodes, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_BototmNodes into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_p);
        return 1;
    }

    /*louncing the kernel*/
    int threads = 1024;                                  /*Thread per blocks on x dir */
    int blocks = nodes/threads +1;                       /*Blocks on the x dir*/

    BCdisp<<<blocks, threads >>>(d_p,d_TopNodes,d_BottomNodes,nodes,disp,yh);

    //gpuErrchk(hipPeekAtLastError()); 
    //gpuErrchk(hipDeviceSynchronize());

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching AddInts!\n";
        return 1;
    }

    cudaStatus = hipMemcpy(p,d_p, sizeof(double) * nodes * 3, hipMemcpyDeviceToHost) ;
    if(cudaStatus!= hipSuccess){
        delete[] p;
        hipFree(d_p);
        std:: cout<<" \n Could not copy d_p back into the cpu !";
        std::cerr << "hipMemcpy device to host failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "AddInts launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }


    hipFree(d_p);
    hipFree(d_TopNodes);
    hipFree(d_BottomNodes);

    return 0;
}
