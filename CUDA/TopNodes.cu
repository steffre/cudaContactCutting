#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include "../Header/functions.h"

__global__ void AddInts(double *p,int *TopNodes, int nodes, double BoundSize, double yh){
    int nodeidx = (blockIdx.x * blockDim.x) + threadIdx.x;

    int pIdx_y = nodeidx * 3 + 1;
    double limit = yh-BoundSize;

    if (nodeidx < nodes){
        /*this is finding all the nodes in p that has the y position higher than yh - BoundSize */
        if (p[pIdx_y] >= (limit)){
            TopNodes[nodeidx] = 1;
        }
        else{
            TopNodes[nodeidx] = 0;
        }
        
    }
}


int topB(int nodes,double ori, double *p,int *TopNodes,double BoundSize, double yh){

    double *d_p;   /* p matrix that has the coordnate of the nodes as pointer*/
    int *d_TopNodes; /* TopNodes pointer to the memory that has top nodes saved*/

    hipError_t cudaStatus;
    cudaStatus = hipMalloc(&d_p,sizeof(double) * nodes * 3);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_p! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_TopNodes,sizeof(int) * nodes);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_p! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_p,p, sizeof(double) * nodes * 3, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_p into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_p);
        return 1;
    }

    /* copy into the device memory */
    /*cudaStatus = hipMemcpy(d_TopNodes,TopNodes, sizeof(int) * nodes, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_p into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_p);
        return 1;
    }*/

    /*louncing the kernel*/
    int threads = 1024;                                   /*Thread per blocks on x dir */
    int blocks = nodes/threads +1;                       /*Blocks on the x dir*/

    AddInts<<<blocks, threads >>>(d_p,d_TopNodes,nodes,BoundSize,yh);

    //gpuErrchk(hipPeekAtLastError()); 
    //gpuErrchk(hipDeviceSynchronize());

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching AddInts!\n";
        return 1;
    }

    cudaStatus = hipMemcpy(p,d_p, sizeof(double) * nodes * 3, hipMemcpyDeviceToHost) ;
    if(cudaStatus!= hipSuccess){
        delete[] p;
        hipFree(d_p);
        std:: cout<<" \n Could not copy d_p back into the cpu !";
        std::cerr << "hipMemcpy device to host failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    
    cudaStatus = hipMemcpy(TopNodes, d_TopNodes, sizeof(int) * nodes, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Failed to copy d_TopNodes from device to host: " << hipGetErrorString(cudaStatus) << "\n";
        hipFree(d_p);
        hipFree(d_TopNodes);
        return 1;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "AddInts launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }


    hipFree(d_p);
    hipFree(d_TopNodes);

    return 0;
}
