#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include "../Header/functions.h"
#include <cmath>

__global__ void col(int nodes, int maxncon, int *color,int *damage,int *Nncon){
    int nodeidx = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    if (nodeidx < nodes){

        color[nodeidx] = 0;
        
        for (int i=0;i<Nncon[nodeidx];i++){
            if (damage [nodeidx * maxncon + i] == 0){
                color[nodeidx] = 1;
            }
        }
    }
}


int colorDamage(int nodes, int *color, int *damage, int maxncon, int *Nncon){

    int *d_color;       /* vector that tells for ech node connection */
    int *d_Nncon;       /* vector that tells for ech node connection */
    int *d_damage;      /* vector that tells for ech spring which one is still active or not */

    hipError_t cudaStatus;
    

    cudaStatus = hipMalloc(&d_color,sizeof(int) * nodes);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_Nncon! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_Nncon,sizeof(int) * nodes);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_Nncon! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_damage,sizeof(int) * nodes * maxncon);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_Nncon! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_color,color, sizeof(int) * nodes, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy color into d_color into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_color);
        return 1;
    }

    cudaStatus = hipMemcpy(d_Nncon,Nncon, sizeof(int) * nodes, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy Nncon into d_Nncon into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_color);
        return 1;
    }

    cudaStatus = hipMemcpy(d_damage,damage, sizeof(int) * nodes * maxncon, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy damage into d_damage into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_damage);
        return 1;
    }

    

    /*louncing the kernel*/
    int threads = 512;                                   /*Thread per blocks on x dir */
    int blocks = (nodes + threads - 1) / threads;   // Ensure that all nodes are covered    

    col<<< blocks, threads >>>(nodes,maxncon,d_color,d_damage,d_Nncon);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "findE kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_color);
        hipFree(d_Nncon);
        hipFree(d_damage);
        return 1;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching findE!\n";
        hipFree(d_color);
        hipFree(d_Nncon);
        hipFree(d_damage);
        
        return 1;
    }
    
    //gpuErrchk(hipPeekAtLastError()); 
    //gpuErrchk(hipDeviceSynchronize());

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching nodCon!\n";
        return 1;
    }

    cudaStatus = hipMemcpy(color, d_color, sizeof(int) * nodes, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        delete[] color;
        hipFree(d_color);
        hipFree(d_Nncon);
        hipFree(d_damage);
        
        std::cerr << "Failed to copy d_damage from device to host: " << hipGetErrorString(cudaStatus) << "\n";
        return 1;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "nodCon launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }


    
    hipFree(d_color);
    hipFree(d_Nncon);
    hipFree(d_damage);
    
    

    return 0;
}
