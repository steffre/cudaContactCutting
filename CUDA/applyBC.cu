#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include "../Header/functions.h"

__global__ void applyBCkernel(double *Fnodes,int *TopNodes, int *BottomNodes, int nodes){
    
    int nodeidx = (blockIdx.x * blockDim.x) + threadIdx.x;

    int pIdx_x = nodeidx * 3 + 0;
    int pIdx_y = nodeidx * 3 + 1;
    

    if (nodeidx < nodes ){

        /*this is finding all the nodes in p that has the y position higher than yh - BoundSize */
        if (TopNodes[nodeidx] == 1){
            Fnodes[pIdx_x] = 0;
            Fnodes[pIdx_y] = 0;
        }
        if(BottomNodes[nodeidx] == 1){
            Fnodes[pIdx_x] = 0;
            Fnodes[pIdx_y] = 0;
        }
        
    }
}


int applyBC(double *Fnodes,int *TopNodes,int *BottomNodes, int nodes){

    double *d_Fnodes;       /* p matrix that has the coordnate of the nodes as pointer */
    int *d_TopNodes;        /* TopNodes pointer to the memory that has top nodes saved */
    int *d_BottomNodes;     /* TopNodes pointer to the memory that has top nodes saved */
    

    hipError_t cudaStatus;
    cudaStatus = hipMalloc(&d_Fnodes,sizeof(double) * nodes * 3);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_p! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_TopNodes,sizeof(int) * nodes);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_TopNodes! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_BottomNodes,sizeof(int) * nodes);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_BottomNodes! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_Fnodes,Fnodes, sizeof(double) * nodes * 3, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_p into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_Fnodes);
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_TopNodes,TopNodes, sizeof(int) * nodes, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_TopNodes into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_TopNodes);
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_BottomNodes,BottomNodes, sizeof(int) * nodes, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_BototmNodes into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_BottomNodes);
        return 1;
    }

    /*louncing the kernel*/
    int threads = 1024;                                   /* Thread per blocks on x dir */
    int blocks = nodes/threads+1;              /* Blocks on the x dir*/

    applyBCkernel<<< blocks, threads >>>(d_Fnodes,d_TopNodes,d_BottomNodes,nodes);

    //gpuErrchk(hipPeekAtLastError()); 
    //gpuErrchk(hipDeviceSynchronize());

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching AddInts!\n";
        return 1;
    }

    cudaStatus = hipMemcpy(Fnodes,d_Fnodes, sizeof(double) * nodes * 3, hipMemcpyDeviceToHost) ;
    if(cudaStatus!= hipSuccess){
        delete[] Fnodes;
        hipFree(d_Fnodes);
        std:: cout<<" \n Could not copy d_p back into the cpu !";
        std::cerr << "hipMemcpy device to host failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "AddInts launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    Fnodes[0] = 0; /* fixing one node in the x direction so the model wont slide */
    Fnodes[1] = 0; /* fixing one node in the x direction so the model wont slide */
    Fnodes[2] = 0; /* fixing one node in the z direction so the model wont slide */

    hipFree(d_Fnodes);
    hipFree(d_TopNodes);
    hipFree(d_BottomNodes);

    return 0;
}
