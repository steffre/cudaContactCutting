#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include "../Header/functions.h"

__global__ void updatePkernel(double *p, double *pnew, int nodes){
    
    int nodeidx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (nodeidx < nodes * 3){

        p[nodeidx]=pnew[nodeidx];
        
    }
}


int updateP(double *p,double *pnew, int nodes){

    double *d_p;            /* Vector of the position of the nodes */
    double *d_pnew;         /* New posiiton after apply force */
    

    hipError_t cudaStatus;
    
    cudaStatus = hipMalloc(&d_p,sizeof(double) * nodes * 3);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_p! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_pnew,sizeof(double) * nodes * 3);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_pnew! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_pnew,pnew, sizeof(double) * nodes * 3, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy pnew into d_pnew into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_pnew);
        return 1;
    }

    cudaStatus = hipMemcpy(d_p,p, sizeof(double) * nodes * 3, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_p into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_p);
        return 1;
    }

   

    /*louncing the kernel*/
    int threads = 1024;                                   /* Thread per blocks on x dir */
    int blocks = nodes*3/threads+1;              /* Blocks on the x dir*/

    updatePkernel<<< blocks, threads >>>(d_p,d_pnew,nodes);

    //gpuErrchk(hipPeekAtLastError()); 
    //gpuErrchk(hipDeviceSynchronize());

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching AddInts!\n";
        return 1;
    }

    cudaStatus = hipMemcpy(p,d_p, sizeof(double) * nodes * 3, hipMemcpyDeviceToHost) ;
    if(cudaStatus!= hipSuccess){
        delete[] p;
        hipFree(d_p);
        std:: cout<<" \n Could not copy d_p back into the cpu !";
        std::cerr << "hipMemcpy device to host failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "AddInts launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }


    hipFree(d_p);
    hipFree(d_pnew);

    return 0;
}
