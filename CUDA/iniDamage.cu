#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include "../Header/functions.h"
#include <cmath>

__global__ void inD(double *p,int nodes, int *Nncon, int *ncon,int maxncon, int *d_damage, double yh, double xl){
    int node = (blockIdx.x * blockDim.x) + threadIdx.x;
    int idxncon = node*maxncon;
    
    if (node < nodes){
        //printf("Nncon[%d]: %d\n", node, Nncon[node]);
        for (int j=0;j<Nncon[node];j++){
            d_damage[idxncon + j ] = 1;
            if ( p[ node * 3 + 0 ]< xl/2 ){
                //printf("Node: %d, X: %f, Y: %f, xl/2: %f, yh/2: %f\n", node, p[node * 3 + 0], p[node * 3 + 1], xl/2, yh/2);
                if ((p[ node*3 + 1]< yh/2 && p[ ncon[ idxncon + j ] * 3 + 1]>= yh/2) ||
                    (p[ node*3 + 1]>= yh/2 && p[ ncon[ idxncon + j ] * 3 + 1]< yh/2)){
                    d_damage[idxncon + j ] = 0;  // Set damage to 1 if conditions are met
                }
            }
        }
    }
}


int iniDamage(int nodes, double *p, int *Nncon, int *ncon, int maxncon, int *damage, double yh, double xl){

    double *d_p;    /* p matrix that has the coordnate of the nodes as pointer */
    int *d_Nncon;   /* vector that tells for ech node how many nodes are connected */
    int *d_ncon;    /* vector that tells which nodes are connected */
    int *d_damage;    /* vector that tell what link to desable */

    std::cout << "yh: " << yh << " xl: " << xl << " nodes: " << nodes << std::endl;

    //std::cout<<"\n nodes size : "<<nodes<<"\n";
    //std::cout<<"\n Max n connection : "<<maxncon<<"\n";

    hipError_t cudaStatus;
    cudaStatus = hipMalloc(&d_p,sizeof(double) * nodes * 3);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_p! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_Nncon,sizeof(int) * nodes);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_Nncon! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_ncon,sizeof(int) * nodes*maxncon);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_ncon! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_damage,sizeof(int) * nodes*maxncon);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_damage! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_p,p, sizeof(double) * nodes * 3, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_p into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_p);
        return 1;
    }

    cudaStatus = hipMemcpy(d_Nncon,Nncon, sizeof(int) * nodes, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy Nncon into d_Nncon into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_Nncon);
        return 1;
    }

    cudaStatus = hipMemcpy(d_ncon,ncon, sizeof(int) * nodes * maxncon, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy ncon into d_ncon into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_ncon);
        return 1;
    }
    

    cudaStatus = hipMemset(d_damage, 0, sizeof(int) * nodes * maxncon);
    if(cudaStatus != hipSuccess){
        std::cerr << "hipMemset failed for d_damage: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    /*louncing the kernel*/
    int threads = 512;                                   /*Thread per blocks on x dir */
    int blocks = (nodes+threads-1)/threads;                        /*Blocks on the x dir*/

    inD<<< blocks, threads >>>(d_p,nodes,d_Nncon,d_ncon,maxncon,d_damage,yh,xl);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "inD kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_p);
        hipFree(d_ncon);
        hipFree(d_Nncon);
        hipFree(d_damage);
        return 1;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching inD!\n";
        hipFree(d_p);
        hipFree(d_ncon);
        hipFree(d_Nncon);
        hipFree(d_damage);
        return 1;
    }
    
    //gpuErrchk(hipPeekAtLastError()); 
    //gpuErrchk(hipDeviceSynchronize());

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching nodCon!\n";
        return 1;
    }

    cudaStatus = hipMemcpy(damage, d_damage, sizeof(int) * nodes*maxncon, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        delete[] p;
        hipFree(d_p);
        hipFree(d_ncon);
        hipFree(d_Nncon);
        hipFree(d_damage);
        std::cerr << "Failed to copy d_damage from device to host: " << hipGetErrorString(cudaStatus) << "\n";
        return 1;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "nodCon launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }


    hipFree(d_p);
    hipFree(d_ncon);
    hipFree(d_Nncon);
    hipFree(d_damage);

    return 0;
}
