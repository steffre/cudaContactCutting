#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include "../Header/functions.h"

__device__ double atomicAdd_double(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}


__global__ void Fcompute(double *p, int *ncon, int *Nncon, double *Fnodes, double *l0,int nodes, int maxncon, double mu, double alpha, int *damage){
    
    int nodeidx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int Node1,Node2;
    double l,sb,fi,fix,fiy,fiz,DX,DY,DZ;
    

    if (nodeidx < nodes){
        /* initialization of the forces */
        Fnodes[ nodeidx * 3 + 0 ] = 0;
        Fnodes[ nodeidx * 3 + 1 ] = 0;
        Fnodes[ nodeidx * 3 + 2 ] = 0;
        
        for (int i=0; i < Nncon[ nodeidx ]; i++){
            Node1 = nodeidx * 3;
            Node2 = ncon[ nodeidx * maxncon + i ] * 3;

            DX = p[ Node1 + 0 ] - p[ Node2 + 0 ];
            DY = p[ Node1 + 1 ] - p[ Node2 + 1 ];
            DZ = p[ Node1 + 2 ] - p[ Node2 + 2 ];

            l = sqrt(DX*DX + DY*DY + DZ*DZ);

            //printf("l value for nodeidx %d: %f\n", nodeidx, l);

            sb = l / l0[ nodeidx * maxncon + i ];
            fi = (- mu * ( pow(sb,alpha) - pow(sb, -(1/2*alpha))))*damage[ nodeidx * maxncon + i ];

            fix = fi/l*DX;
            fiy = fi/l*DY;
            fiz = fi/l*DZ;

            atomicAdd_double(&Fnodes[nodeidx * 3 + 0], fix);
            atomicAdd_double(&Fnodes[nodeidx * 3 + 1], fiy);
            atomicAdd_double(&Fnodes[nodeidx * 3 + 2], fiz);

        }
        
        
    }
}


int assemblyForces(int nodes,int nc, double *Fnodes,int *ncon, int *Nncon, int maxncon, double *p,double mu, double alpha, double *l0, int *damage){

    double *d_p;        /* p matrix that has the coordnate of the nodes as pointer*/
    int *d_ncon;        /* nocn contain the nodes that are connected on the single node*/
    int *d_Nncon;       /* Nncon is a vector that contain the number of the nodes that are connected to aspecific node*/
    double *d_Fnodes;      /* Fnodes contains the force in x y z direction on each single node*/
    double *d_l0;       /* this contain the sigle spring lenght base on the ndoeal connection same size of ncon*/
    int *d_damage;       /* damage tanke in consideration the damaged springs*/
    

    

    hipError_t cudaStatus;
    cudaStatus = hipMalloc(&d_p,sizeof(double) * nodes * 3);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_p! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_ncon,sizeof(int) * nodes * maxncon);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_ncon! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_damage,sizeof(int) * nodes * maxncon);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_damage! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_Nncon,sizeof(int) * nodes);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_Nncon! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_Fnodes,sizeof(double) * nodes * 3);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_Fnodes! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_l0,sizeof(double) * nodes * maxncon);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_l0! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_p,p, sizeof(double) * nodes * 3, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_p into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_p);
        return 1;
    }

    cudaStatus = hipMemcpy(d_ncon,ncon, sizeof(int) * nodes * maxncon, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy ncon into d_ncon into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_ncon);
        return 1;
    }

    cudaStatus = hipMemcpy(d_damage,damage, sizeof(int) * nodes * maxncon, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy damage into d_damage into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_damage);
        return 1;
    }

    cudaStatus = hipMemcpy(d_Nncon,Nncon, sizeof(int) * nodes, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy Nncon into d_Nncon into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_Nncon);
        return 1;
    }

    cudaStatus = hipMemcpy(d_Fnodes,Fnodes, sizeof(double) * nodes * 3, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy Fnodes into d_Fnodes into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_Fnodes);
        return 1;
    }

    cudaStatus = hipMemcpy(d_l0,l0, sizeof(double) * nodes * maxncon, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy l0 into d_l0 into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_l0);
        return 1;
    }

    /* louncing the kernel */
    int threads = 512;                                   /* Thread per blocks on x dir */
    int blocks = (nodes+threads-1)/threads;              /* Blocks on the x dir*/

    Fcompute<<<blocks, threads >>>(d_p,d_ncon,d_Nncon,d_Fnodes,d_l0,nodes,maxncon,mu,alpha,d_damage);

    //gpuErrchk(hipPeekAtLastError()); 
    //gpuErrchk(hipDeviceSynchronize());

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching AddInts!\n";
        return 1;
    }

    cudaStatus = hipMemcpy(Fnodes,d_Fnodes, sizeof(double) * nodes * 3, hipMemcpyDeviceToHost) ;
    if(cudaStatus!= hipSuccess){
        delete[] Fnodes;
        hipFree(d_Fnodes);
        std:: cout<<" \n Could not copy d_p back into the cpu !";
        std::cerr << "hipMemcpy device to host failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "AddInts launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }


    hipFree(d_p);
    hipFree(d_ncon);
    hipFree(d_Nncon);
    hipFree(d_Fnodes);
    hipFree(d_l0);
    hipFree(d_damage);

    return 0;
}
