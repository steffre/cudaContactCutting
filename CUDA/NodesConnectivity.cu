#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include "../Header/functions.h"
#include <cmath>

__global__ void nodCon(double *p,int nodes, double *l0, int *Nncon, int *ncon, int *c,int maxncon,int nc){
    int node = (blockIdx.x * blockDim.x) + threadIdx.x;
    int nodex = node * 3 + 0;
    int nodey = node * 3 + 1;
    int nodez = node * 3 + 2;
    
    int idxncon = node*maxncon;
    int k=0,node2,node2x,node2y,node2z;

    double xi,xj,yi,yj,zi,zj;
    
    if (node < nodes){
        /* in this soubroutine we need to pass by all the other nodes and calculate the l0 
        for the one that are conected and save the noide that is connected 
        and the how many nodes are connected */
        for (int j=0;j<nc;j++){
            
            if ( node == c[ j * 2 + 0 ] ){
                node2 = c[ j * 2 + 1 ];
                node2x = node2 * 3;
                node2y = node2 * 3 + 1;
                node2z = node2 * 3 + 2;

                xi = p[ nodex ];
                yi = p[ nodey ];
                zi = p[ nodez ];
                xj = p[ node2x ];
                yj = p[ node2y ];
                zj = p[ node2z ];

                l0 [ idxncon + k ] = pow( (xi-xj)*(xi-xj)+(yi-yj)*(yi-yj)+(zi-zj)*(zi-zj) , 0.5 );
                ncon [ idxncon + k ] = node2;
                k++;

                //printf("Thread %d: \t node: %d, \t l0[idxncon + k]: %f,\t  k: %d,\t j * 2: %d, \t c[j * 2 + 1]: %d\n", threadIdx.x, node, l0[idxncon + k-1],k,j*2, c[j * 2 + 1]);
                

            }
            if ( node == c [ j * 2 + 1 ] ){
                //printf("Thread %d: \t node: %d, \t c[j * 2]: %d,\t j * 2: %d, \t c[j * 2 + 1]: %d\n", threadIdx.x, node, c[j * 2],j*2, c[j * 2 + 1]);
                node2 = c [ j * 2 ];
                node2x = node2 * 3;
                node2y = node2 * 3 + 1;
                node2z = node2 * 3 + 2;

                xi = p[ nodex ];
                yi = p[ nodey ];
                zi = p[ nodez ];
                xj = p[ node2x ];
                yj = p[ node2y ];
                zj = p[ node2z ];

                l0 [ idxncon + k ] = pow( (xi-xj)*(xi-xj)+(yi-yj)*(yi-yj)+(zi-zj)*(zi-zj) , 0.5 );
                ncon [ idxncon + k ] = node2;
                k++;

                //printf("Thread %d: \t node: %d, \t l0[idxncon + k]: %f,\t  k: %d,\t j * 2: %d, \t c[j * 2]: %d\n", threadIdx.x, node, l0[idxncon + k-1],k,j*2, c[j * 2 ]);
                
            }
        }
        Nncon[node] = k;
        k=0;
    }
}


int NodalConnection(int nodes, int nc,  int *c, double *p, double *l0, int *Nncon, int *ncon, int maxncon){

    double *d_p;    /* p matrix that has the coordnate of the nodes as pointer */
    int *d_c;       /* connectivity matrix device pointer */
    double *d_l0;   /* l0 initial lenght of the springs */
    int *d_Nncon;   /* vector that tells for ech node how many nodes are connected */
    int *d_ncon;    /* vector that tells which nodes are connected */


    //std::cout<<"\n nodes size : "<<nodes<<"\n";
    //std::cout<<"\n Max n connection : "<<maxncon<<"\n";

    hipError_t cudaStatus;
    cudaStatus = hipMalloc(&d_p,sizeof(double) * nodes * 3);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_p! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_c,sizeof(int) * nc * 2);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_c! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_l0,sizeof(double) * nodes*maxncon);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_l0! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_Nncon,sizeof(int) * nodes);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_Nncon! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_ncon,sizeof(int) * nodes*maxncon);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_ncon! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_p,p, sizeof(double) * nodes * 3, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_p into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_p);
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_c,c, sizeof(int) * nc * 2, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy c into d_c into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_c);
        return 1;
    }

    cudaStatus = hipMemset(d_l0, 0, sizeof(double) * nodes * maxncon);
    if(cudaStatus != hipSuccess){
        std::cerr << "hipMemset failed for d_l0: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMemset(d_Nncon, 0, sizeof(int) * nodes);
    if(cudaStatus != hipSuccess){
        std::cerr << "hipMemset failed for d_Nncon: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMemset(d_ncon, 0, sizeof(int) * nodes * maxncon);
    if(cudaStatus != hipSuccess){
        std::cerr << "hipMemset failed for d_ncon: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }


    /*louncing the kernel*/
    int threads = 512;                                   /*Thread per blocks on x dir */
    int blocks = (nodes+threads-1)/threads;                        /*Blocks on the x dir*/

    nodCon<<< blocks, threads >>>(d_p,nodes,d_l0,d_Nncon,d_ncon,d_c,maxncon,nc);

    //gpuErrchk(hipPeekAtLastError()); 
    //gpuErrchk(hipDeviceSynchronize());

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching nodCon!\n";
        return 1;
    }
    //std::cerr << "pre copy d_Nncon " << Nncon[0] << "\n";
    cudaStatus = hipMemcpy(Nncon, d_Nncon, sizeof(int) * nodes, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        delete[] p;
        hipFree(d_p);
        hipFree(d_c);
        hipFree(d_ncon);
        hipFree(d_Nncon);
        hipFree(d_l0);
        std::cerr << "Failed to copy d_Nncon from device to host: " << hipGetErrorString(cudaStatus) << "\n";
        return 1;
    }
    //std::cerr << "after copy d_Nncon " << Nncon[0] << "\n";

    cudaStatus = hipMemcpy(ncon, d_ncon, sizeof(int) * nodes*maxncon, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        delete[] p;
        hipFree(d_p);
        hipFree(d_c);
        hipFree(d_ncon);
        hipFree(d_Nncon);
        hipFree(d_l0);
        std::cerr << "Failed to copy d_ncon from device to host: " << hipGetErrorString(cudaStatus) << "\n";
        return 1;
    }

    cudaStatus = hipMemcpy(l0, d_l0, sizeof(double) * nodes*maxncon, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        delete[] p;
        hipFree(d_c);
        hipFree(d_p);
        hipFree(d_ncon);
        hipFree(d_Nncon);
        hipFree(d_l0);
        std::cerr << "Failed to copy d_l0 from device to host: " << hipGetErrorString(cudaStatus) << "\n";
        return 1;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "nodCon launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }


    hipFree(d_p);
    hipFree(d_ncon);
    hipFree(d_Nncon);
    hipFree(d_l0);
    hipFree(d_c);

    return 0;
}
