#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include "../Header/functions.h"

__global__ void AddInts(double *p,int *BottomNodes, int nodes, double BoundSize){
    int nodeidx = (blockIdx.x * blockDim.x) + threadIdx.x;

    int pIdx_y = nodeidx * 3 + 1;
    

    if (nodeidx < nodes){
        /*this is finding all the nodes in p that has the y position higher than yh - BoundSize */
        if (p[pIdx_y] <= (BoundSize)){
            BottomNodes[nodeidx] = 1;
        }
        else{
            BottomNodes[nodeidx] = 0;
        }
        
    }
}


int bottomB(int nodes,double ori, double *p,int *BottomNodes,double BoundSize){

    double *d_p;   /* p matrix that has the coordnate of the nodes as pointer*/
    int *d_BottomNodes; /* TopNodes pointer to the memory that has top nodes saved*/

    hipError_t cudaStatus;
    cudaStatus = hipMalloc(&d_p,sizeof(double) * nodes * 3);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_p! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_BottomNodes,sizeof(int) * nodes);
    if(cudaStatus!= hipSuccess){
        std::cout<<"\n Malloc d_p! nope ... ";
        std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    /* copy into the device memory */
    cudaStatus = hipMemcpy(d_p,p, sizeof(double) * nodes * 3, hipMemcpyHostToDevice );
    if(cudaStatus!= hipSuccess){
        std::cout << "\n Could not copy p into d_p into the device !";
        std::cerr << "hipMemcpy host to device failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_p);
        return 1;
    }

    /*louncing the kernel*/
    int threads = 1024;                                   /*Thread per blocks on x dir */
    int blocks = nodes/threads +1;                       /*Blocks on the x dir*/

    AddInts<<<blocks, threads >>>(d_p,d_BottomNodes,nodes,BoundSize);

    //gpuErrchk(hipPeekAtLastError()); 
    //gpuErrchk(hipDeviceSynchronize());

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching AddInts!\n";
        return 1;
    }

    cudaStatus = hipMemcpy(p,d_p, sizeof(double) * nodes * 3, hipMemcpyDeviceToHost) ;
    if(cudaStatus!= hipSuccess){
        delete[] p;
        hipFree(d_p);
        std:: cout<<" \n Could not copy d_p back into the cpu !";
        std::cerr << "hipMemcpy device to host failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    
    cudaStatus = hipMemcpy(BottomNodes, d_BottomNodes, sizeof(int) * nodes, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "Failed to copy d_TopNodes from device to host: " << hipGetErrorString(cudaStatus) << "\n";
        hipFree(d_p);
        hipFree(d_BottomNodes);
        return 1;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "AddInts launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }


    hipFree(d_p);
    hipFree(d_BottomNodes);

    return 0;
}
